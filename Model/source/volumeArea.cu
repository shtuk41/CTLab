#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include <vecMath.cuh>

#include <iostream>
#include <vector>
#include <glm/glm.hpp>

#include <scanObject.h>

#ifdef __INTELLISENSE__
// Dummy definitions to suppress IntelliSense errors
typedef struct { unsigned int x, y, z; } dim3;
extern dim3 threadIdx;
extern dim3 blockIdx;
extern dim3 blockDim;
extern dim3 gridDim;
#endif


__global__ void findInnerPointsKernel(const float3* d_mesh, int numberOfTriangles, const float3*d_areaPoints, bool*d_includeFlags, size_t numberOfPoints) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= numberOfPoints)
		return;

	float3 p = d_areaPoints[idx];

	float totalSolidAngle = 0.0f;

	for (int t = 0; t < numberOfTriangles; ++t) 
	{
		float3 a = vecSub(d_mesh[t * 3 + 0], p);
		float3 b = vecSub(d_mesh[t * 3 + 1], p);
		float3 c = vecSub(d_mesh[t * 3 + 2], p);

		float la = vecLength(a);
		float lb = vecLength(b);
		float lc = vecLength(c);

		float numerator = vecDot(a, vecCross(b, c));

		float denominator = la * lb * lc
			+ vecDot(a, b) * lc
			+ vecDot(b, c) * la
			+ vecDot(c, a) * lb;

		float omega = 2.0f * atan2f(numerator, denominator);
		totalSolidAngle += omega;

	}

	const float epsilon = 1e-2f;
	const float four_pi = 4.0f * HIP_PI_F;

	d_includeFlags[idx] = fabsf(totalSolidAngle - four_pi) < epsilon;
}

void getPointsInsideObjectCudaKernelLouncher(const ScanObject& obj, ZArray* scanBox, std::vector<glm::vec3> & internalPoints)
{
	unsigned int numberOfTriangles = obj.GetNumberOfTriangles();
	glm::vec4* ptr = obj.GetTrianglesWithOffset();
	std::vector<float3> meshPtr(numberOfTriangles * 3);

	for (unsigned int ii = 0; ii < numberOfTriangles; ii++) 
	{
		meshPtr[ii * 3 + 0] = make_float3(ptr[ii * 3 + 0].x, ptr[ii * 3 + 0].y, ptr[ii * 3 + 0].z);
		meshPtr[ii * 3 + 1] = make_float3(ptr[ii * 3 + 1].x, ptr[ii * 3 + 1].y, ptr[ii * 3 + 1].z);
		meshPtr[ii * 3 + 2] = make_float3(ptr[ii * 3 + 2].x, ptr[ii * 3 + 2].y, ptr[ii * 3 + 2].z);
	}

	std::vector<float3> areaPoints(nVoxelsZ * nVoxelsY * nVoxelsX);

	for (int h = 0; h < nVoxelsZ; h++)
	{
		for (int w = 0; w < nVoxelsY; w++)
		{
			for (int d = 0; d < nVoxelsX; d++)
			{
				glm::vec3 p = (*scanBox)[h][w][d].first;

				int index = ((h * nVoxelsY + w) * nVoxelsX + d);

				areaPoints[index] = make_float3(p.x, p.y, p.z);
			}
		}
	}

	size_t totalVoxes = nVoxelsZ * nVoxelsY * nVoxelsX;

	float3* d_mesh;
	hipMalloc(&d_mesh, numberOfTriangles * 3 * sizeof(float3));
	hipMemcpy(d_mesh, meshPtr.data(), numberOfTriangles * 3 * sizeof(float3), hipMemcpyHostToDevice);

	float3* d_areaPoints;
	hipMalloc(&d_areaPoints, totalVoxes * sizeof(float3));
	hipMemcpy(d_areaPoints, areaPoints.data(), totalVoxes * sizeof(float3), hipMemcpyHostToDevice);

	bool* d_includeFlag;
	hipMalloc(&d_includeFlag, totalVoxes * sizeof(bool));
	hipMemset(d_includeFlag, 0, totalVoxes);

	int threadsPerBlock = 256;
	int blocks = static_cast<int>((totalVoxes + threadsPerBlock - 1) / threadsPerBlock);
	findInnerPointsKernel<<<blocks, threadsPerBlock>>>(d_mesh, numberOfTriangles, d_areaPoints, d_includeFlag, totalVoxes);

	hipError_t err = hipDeviceSynchronize();

	if (err != hipSuccess) 
	{
		std::cerr << "CUDA error after findInnerPointsKernel launch: " << hipGetErrorString(err) << std::endl;
	}

	err = hipGetLastError();
	if (err != hipSuccess) {
		std::cerr << "CUDA kernel findInnerPointsKernel error: " << hipGetErrorString(err) << std::endl;
		return;
	}

	bool* h_includeFlag = new bool[totalVoxes];
	hipMemcpy(h_includeFlag, d_includeFlag, totalVoxes * sizeof(bool), hipMemcpyDeviceToHost);

	for (int ii = 0; ii < totalVoxes; ii++)
	{
		if (h_includeFlag[ii])
		{
			int d = ii % nVoxelsX;
			int w = (ii / nVoxelsX) % nVoxelsY;
			int h = ii / (nVoxelsX * nVoxelsY);

			internalPoints.push_back((*scanBox)[h][w][d].first);
		}
	}

	delete[] h_includeFlag;

	hipFree(d_mesh);
	hipFree(d_areaPoints);
	hipFree(d_includeFlag);

}

