#include <hip/hip_runtime.h>

#ifdef __INTELLISENSE__
// Dummy definitions to suppress IntelliSense errors
typedef struct { unsigned int x, y, z; } dim3;
extern dim3 threadIdx;
extern dim3 blockIdx;
extern dim3 blockDim;
extern dim3 gridDim;
#endif

void myCudaKernelLauncher()
{

}

__global__ void myKernel(int* data) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // do work per thread
}