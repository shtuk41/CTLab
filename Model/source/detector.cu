
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

#include <opencv2/opencv.hpp>

#include <iostream>
#include <vector>
#include <glm/glm.hpp>

#include <scanObject.h>

#ifdef __INTELLISENSE__
// Dummy definitions to suppress IntelliSense errors
typedef struct { unsigned int x, y, z; } dim3;
extern dim3 threadIdx;
extern dim3 blockIdx;
extern dim3 blockDim;
extern dim3 gridDim;
#endif



void getPixelsPyramidMethodKernelLouncher(const glm::vec3& center, const std::vector<glm::vec3>& areaPoints, cv::Mat& detectorImage)
{

}


