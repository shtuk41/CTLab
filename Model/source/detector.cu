#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include <vecMath.cuh>


#include <opencv2/opencv.hpp>

#include <iostream>
#include <vector>
#include <glm/glm.hpp>

#include <scanObject.h>
#include <detector.h>


#ifdef __INTELLISENSE__
// Dummy definitions to suppress IntelliSense errors
typedef struct { unsigned int x, y, z; } dim3;
extern dim3 threadIdx;
extern dim3 blockIdx;
extern dim3 blockDim;
extern dim3 gridDim;
#endif


__global__ void getPixelPyramidMethodKernel(const float3* detectorPixels, size_t numberPixels, float3 sourceCenter, const float3* d_areaPoints, size_t numberOfPoints, unsigned short* d_pixelValue)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= numberPixels)
		return;

	float3 xV = vecSub(detectorPixels[idx], sourceCenter);
	float h = vecLength(xV);
	
	if (h < 1e-6f) 
		return; // avoid zero length

	xV = vecNormalize(xV);

	// Build orthonormal frame
	float3 arbitrary = (fabs(vecDot(xV, make_float3(0, 0, 1))) < 0.99f) ? make_float3(0, 0, 1) : make_float3(1, 0, 0);
	float3 yV = vecNormalize(vecCross(xV, arbitrary));
	float3 zV = vecCross(xV, yV); // right-handed frame

	//&%&%&%&%&%&%&%&%&xVoxelPitch!!!!!!!!!!!!!!!!!!!!!!!!!! fix this
	float half_side = static_cast<float>(0.1 / 2.0);

	ushort currentValue = 0;

	for (int t = 0; t < numberOfPoints; ++t)
	{
		float3 apexToPoint = vecSub(d_areaPoints[t], sourceCenter);

		float px = vecDot(apexToPoint, xV);  // along pyramid axis (height)
		float py = vecDot(apexToPoint, yV);  // lateral
		float pz = vecDot(apexToPoint, zV);  // lateral

		if (px >= 0 && px <= h &&
			fabs(py) <= half_side &&
			fabs(pz) <= half_side)
		{
			currentValue += 1;
		}
	}

	d_pixelValue[idx] = currentValue;
}


void getPixelsPyramidMethodKernelLouncher(const Detector * const detector, const glm::vec3& sourceCenter, const std::vector<glm::vec3>& areaPoints, cv::Mat& detectorImage)
{
	size_t numberPixels = nDetectorResZ * nDetectorResY;

	std::vector<float3> h_detectorPixels(numberPixels);

	for (int jj = 0; jj < nDetectorResZ; jj++)
	{
		for (int ii = 0; ii < nDetectorResY; ii++)
		{
			//std::cout << jj << " " << ii << '\n';
			glm::vec3 pixel = detector->getPixel(jj, ii);

			h_detectorPixels[jj * nDetectorResY + ii] = make_float3(pixel.x, pixel.y, pixel.z);
		}
	}

	float3* d_detectorPoints;
	hipMalloc(&d_detectorPoints, numberPixels * sizeof(float3));
	hipMemcpy(d_detectorPoints, h_detectorPixels.data(), h_detectorPixels.size() * sizeof(float3), hipMemcpyHostToDevice);


	float3 h_sCenter = make_float3(sourceCenter.x, sourceCenter.y, sourceCenter.z);

	std::vector<float3> h_areaPoints(areaPoints.size());

	for (int ii = 0; ii < areaPoints.size(); ii++)
	{
		h_areaPoints[ii] = make_float3(areaPoints[ii].x, areaPoints[ii].y, areaPoints[ii].z);
	}

	float3* d_areaPoints;
	hipMalloc(&d_areaPoints, h_areaPoints.size() * sizeof(float3));
	hipMemcpy(d_areaPoints, h_areaPoints.data(), h_areaPoints.size() * sizeof(float3), hipMemcpyHostToDevice);

	unsigned short* d_pixelValues;
	hipMalloc(&d_pixelValues, numberPixels * sizeof(unsigned short));
	hipMemset(d_pixelValues, 0, numberPixels * sizeof(unsigned short));

	unsigned int threadsPerBlock = 256;
	unsigned int blocks = static_cast<unsigned int>((numberPixels + threadsPerBlock - 1) / threadsPerBlock);
	getPixelPyramidMethodKernel<<<blocks, threadsPerBlock>>> (d_detectorPoints, numberPixels, h_sCenter, d_areaPoints, h_areaPoints.size(), d_pixelValues);

	hipError_t err = hipDeviceSynchronize();

	if (err != hipSuccess)
	{
		std::cerr << "CUDA error after getPixelPyramidMethodKernel lounch: " << hipGetErrorString(err) << std::endl;
	}

	err = hipGetLastError();
	if (err != hipSuccess) {
		std::cerr << "CUDA kernel getPixelPyramidMethodKernel error: " << hipGetErrorString(err) << std::endl;
		return;
	}

	ushort* h_pixelValues = new ushort[numberPixels];

	hipMemcpy(h_pixelValues, d_pixelValues, numberPixels * sizeof(unsigned short), hipMemcpyDeviceToHost);

	ushort maxValue = 1;

	unsigned short* iter = h_pixelValues;

	for (int ii = 0; ii < numberPixels; ii++, iter++)
	{
		if (*iter > maxValue)
			maxValue = *iter;
	}

	for (int jj = 0; jj < nDetectorResZ; jj++)
	{
		for (int ii = 0; ii < nDetectorResY; ii++)
		{
			detectorImage.at<ushort>(jj, ii) = static_cast<ushort>((1.0f - (float(h_pixelValues[jj * nDetectorResY + ii]) / maxValue)) * 65535);
		}
	}

	delete[] h_pixelValues;
	hipFree(d_areaPoints);
	hipFree(d_pixelValues);
	hipFree(d_detectorPoints);
}


